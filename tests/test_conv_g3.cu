#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "../src/memory/types.hh"
#include "../src/ops/vect-sigmoid.hh"
#include "../src/ops/log-softmax.hh"
#include "../src/ops/conv2d.hh"
#include "../src/ops/variable.hh"
#include "../src/ops/input.hh"
#include "../src/ops/ops-builder.hh"
#include "../src/ops/graph.hh"
#include "../src/api/layers.hh"
#include "../src/api/cost.hh"
#include "../src/ops/conv2d-transpose.hh"

#include <tocha/tensor.hh>
#include "../src/datasets/mnist.hh"
#include "../src/memory/alloc.hh"

#define BATCH 64

int main(int argc, char** argv)
{

    if (argc < 4)
    {
        std::cerr << "Invalid number of arguments\n";
        return 1;
    }

    auto weights = tocha::Tensors::load(argv[2]);
    auto x = reinterpret_cast<dbl_t*>(weights.arr()[0].data);
    auto w = reinterpret_cast<dbl_t*>(weights.arr()[1].data);

    auto& builder = ops::OpsBuilder::instance();

    auto x_node = builder.input(ops::Shape({BATCH, 32, 32, 64}));
    auto w_node = builder.input(ops::Shape({5, 5, 3, 64}));
    const int strides[2] = {2, 2};

    const int out_size[4] = {BATCH, 64, 64, 3};
    auto y_node = builder.conv2d_transpose(x_node, w_node, out_size, strides);

    auto& graph = ops::Graph::instance();


    tocha::Tensors out;
    out.add(tocha::Tensor::f32(BATCH, 64, 64, 3));
    dbl_t* y_out = reinterpret_cast<dbl_t*>(out.arr()[0].data);


    graph.run({y_node},
	      {
                  {x_node, {x, ops::Shape({BATCH, 32, 32, 64})}},
                  {w_node, {w, ops::Shape({5, 5, 3, 64})}}
              },
	      {y_out});

    out.save(argv[3]);
}
