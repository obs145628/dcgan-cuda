#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "../src/memory/types.hh"
#include "../src/ops/vect-sigmoid.hh"
#include "../src/ops/log-softmax.hh"
#include "../src/ops/conv2d.hh"
#include "../src/ops/variable.hh"
#include "../src/ops/input.hh"
#include "../src/ops/ops-builder.hh"
#include "../src/ops/graph.hh"
#include "../src/api/layers.hh"
#include "../src/api/cost.hh"
#include "../src/ops/reshape.hh"
#include "../src/ops/mse.hh"
#include "../src/ops/conv2d-transpose.hh"

#include <tocha/tensor.hh>
#include "../src/datasets/mnist.hh"
#include "../src/memory/alloc.hh"

#define BATCH 64

int main(int argc, char** argv)
{

    if (argc < 4)
    {
        std::cerr << "Invalid number of arguments\n";
        return 1;
    }

    
    
    auto weights = tocha::Tensors::load(argv[2]);
    auto x = reinterpret_cast<dbl_t*>(weights.arr()[0].data);
    auto w = reinterpret_cast<dbl_t*>(weights.arr()[1].data);
    auto y = reinterpret_cast<dbl_t*>(weights.arr()[2].data);

    auto& builder = ops::OpsBuilder::instance();
    auto& graph = ops::Graph::instance();
    
    auto x_node = builder.input(ops::Shape({BATCH, 8, 8, 256}));
    auto w_node = builder.input(ops::Shape({5, 5, 128, 256}));
    auto y_node = builder.input(ops::Shape({BATCH, 16 * 16 * 128}));
    const int strides[2] = {2, 2};
    
    const int out_size[4] = {BATCH, 16, 16, 128};
    ops::Op* yh_node = builder.conv2d_transpose(x_node, w_node, out_size, strides);
    yh_node = builder.reshape(yh_node, ops::Shape({BATCH, 16 * 16 * 128}));

    auto mse_node = builder.mse(y_node, yh_node);
    auto dx_node = graph.gradient(mse_node, x_node);


    tocha::Tensors out;
    out.add(tocha::Tensor::f32(BATCH, 8, 8, 256));
    dbl_t* dx_out = reinterpret_cast<dbl_t*>(out.arr()[0].data);


    graph.run({dx_node},
	      {
                  {x_node, {x, ops::Shape({BATCH, 8, 8, 256})}},
                  {w_node, {w, ops::Shape({5, 5, 128, 256})}},
                  {y_node, {y, ops::Shape({BATCH, 16 * 16 * 128})}},
              },
	      {dx_out});

    out.save(argv[3]);
}
