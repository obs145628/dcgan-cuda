#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "../src/memory/types.hh"
#include "../src/ops/vect-tanh.hh"
#include "../src/ops/softmax.hh"
#include "../src/ops/variable.hh"
#include "../src/ops/input.hh"
#include "../src/ops/ops-builder.hh"
#include "../src/ops/mse.hh"
#include "../src/ops/graph.hh"
#include "../src/api/layers.hh"
#include "../src/api/cost.hh"

#include <tocha/tensor.hh>
#include "../src/datasets/mnist.hh"
#include "../src/memory/alloc.hh"

#include "big_mat.hh"

int main(int argc, char** argv)
{

    if (argc != 2)
    {
        std::cerr << "Invalid number of arguments\n";
        return 1;
    }

    const int size = sizeof(a) / (3 * sizeof(dbl_t));

    auto& graph = ops::Graph::instance();

    auto& builder = ops::OpsBuilder::instance();

    auto x_node = builder.input(ops::Shape({size, 3}));
    auto y_node = builder.input(ops::Shape({size, 3}));
    auto y_hat_node = builder.vect_tanh(x_node);
    
    auto loss_node = builder.mse(y_node, y_hat_node);
    auto dx_node = graph.gradient(loss_node, x_node);
    auto dy_hat_node = graph.gradient(loss_node, y_hat_node);


    tocha::Tensors out;
    out.add(tocha::Tensor::f32(size, 3));
    dbl_t* y_hat = reinterpret_cast<dbl_t*>(out.arr()[0].data);
    out.add(tocha::Tensor::f32(1));
    dbl_t* loss = reinterpret_cast<dbl_t*>(out.arr()[1].data);
    out.add(tocha::Tensor::f32(size, 3));
    dbl_t* dx = reinterpret_cast<dbl_t*>(out.arr()[2].data);
    out.add(tocha::Tensor::f32(size, 3));
    dbl_t* dy_hat = reinterpret_cast<dbl_t*>(out.arr()[3].data);

    graph.run({y_hat_node, loss_node, dx_node, dy_hat_node},
              {{x_node, {a, ops::Shape({size, 3})}},
                  {y_node, {a, ops::Shape({size, 3})}}},
	      {y_hat, loss, dx, dy_hat});
    
    
    out.save(argv[1]);
}
