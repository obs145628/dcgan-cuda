#include "hip/hip_runtime.h"
#include "softmax.hh"
#include "../runtime/node.hh"



namespace gpu
{

    namespace
    {
        constexpr std::size_t BLOCK_SIZE = 512;

        __global__
        void softmax1(const dbl_t* x, dbl_t* y,
                      std::size_t rows, std::size_t cols) //8ms
        {
            __shared__ dbl_t partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto row = blockIdx.x;
            auto col = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            dbl_t init = 1e-30;
            for (std::size_t i = col; i < cols; i += step)
                init = max(x[row * cols + i], init);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] = max(partial[col], partial[col + s]);

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile dbl_t* vpartial = partial;
            if (col < 32)
            {
                vpartial[col] = max(vpartial[col], vpartial[col + 32]);
                vpartial[col] = max(vpartial[col], vpartial[col + 16]);
                vpartial[col] = max(vpartial[col], vpartial[col + 8]);
                vpartial[col] = max(vpartial[col], vpartial[col + 4]);
                vpartial[col] = max(vpartial[col],vpartial[col + 2]);
                vpartial[col] = max(vpartial[col], vpartial[col + 1]);
            }
            
            __syncthreads();
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] = exp(x[row * cols + i] - partial[0]);

            __syncthreads();

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += y[row * cols + i];
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }


            __syncthreads();
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] /= partial[0];
        }

        __global__
        void log_softmax1(const dbl_t* x, dbl_t* y,
                      std::size_t rows, std::size_t cols) //8ms
        {
            __shared__ dbl_t partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto row = blockIdx.x;
            auto col = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            dbl_t init = 1e-30;
            for (std::size_t i = col; i < cols; i += step)
                init = max(x[row * cols + i], init);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] = max(partial[col], partial[col + s]);

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile dbl_t* vpartial = partial;
            if (col < 32)
            {
                vpartial[col] = max(vpartial[col], vpartial[col + 32]);
                vpartial[col] = max(vpartial[col], vpartial[col + 16]);
                vpartial[col] = max(vpartial[col], vpartial[col + 8]);
                vpartial[col] = max(vpartial[col], vpartial[col + 4]);
                vpartial[col] = max(vpartial[col],vpartial[col + 2]);
                vpartial[col] = max(vpartial[col], vpartial[col + 1]);
            }
            
            __syncthreads();

            dbl_t max_x = partial[0];

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += exp(x[row * cols + i] - max_x);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }


            __syncthreads();

            dbl_t logsum = max_x + std::log(partial[0]);
            
            for (std::size_t i = col; i < cols; i += step)
                y[row * cols + i] = x[row * cols + i] - logsum;
        }

        __global__
        void softmax_lcost1(const dbl_t* y, const dbl_t* x, dbl_t* out,
                            std::size_t rows, std::size_t cols)
        {
            __shared__ dbl_t partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto row = blockIdx.x;
            auto col = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            dbl_t init = 1e-30;
            for (std::size_t i = col; i < cols; i += step)
                init = max(x[row * cols + i], init);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] = max(partial[col], partial[col + s]);

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile dbl_t* vpartial = partial;
            if (col < 32)
            {
                vpartial[col] = max(vpartial[col], vpartial[col + 32]);
                vpartial[col] = max(vpartial[col], vpartial[col + 16]);
                vpartial[col] = max(vpartial[col], vpartial[col + 8]);
                vpartial[col] = max(vpartial[col], vpartial[col + 4]);
                vpartial[col] = max(vpartial[col],vpartial[col + 2]);
                vpartial[col] = max(vpartial[col], vpartial[col + 1]);
            }
            
            __syncthreads();

            dbl_t max_x = partial[0];

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += exp(x[row * cols + i] - max_x);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }


            __syncthreads();

            dbl_t logsum = max_x + std::log(partial[0]);

            init = 0;
            for (std::size_t i = col; i < cols; i += step)
                init += y[row * cols + i] * (x[row * cols + i] - logsum);
        
            partial[col] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (col < s)
                    partial[col] += partial[col + s];

                __syncthreads();
            }

            if (col < 32)
            {
                vpartial[col] += vpartial[col + 32];
                vpartial[col] += vpartial[col + 16];
                vpartial[col] += vpartial[col + 8];
                vpartial[col] += vpartial[col + 4];
                vpartial[col] += vpartial[col + 2];
                vpartial[col] += vpartial[col + 1];
            }
            
            if (col == 0)
                out[row] = partial[0];
        }

        __global__
        void softmax_reduce(const dbl_t* x, dbl_t* y, std::size_t len) //8ms
        {
            __shared__ dbl_t partial[2 * BLOCK_SIZE];

            //load all elements of the array in shared memory
            auto i = threadIdx.x;
            std::size_t step = BLOCK_SIZE;

            dbl_t init = 0;
            for (std::size_t j = i; j < len; j += step)
                init += x[j];
        
            partial[i] = init;
            __syncthreads();

            for (std::size_t s = BLOCK_SIZE / 2; s > 32; s >>= 1)
            {
                if (i < s)
                    partial[i] += partial[i + s];

                __syncthreads();
            }

            //if not volatile, must use __synctthreads again, why ?
            volatile dbl_t* vpartial = partial;
            if (i < 32)
            {
                vpartial[i] += vpartial[i + 32];
                vpartial[i] += vpartial[i + 16];
                vpartial[i] += vpartial[i + 8];
                vpartial[i] += vpartial[i + 4];
                vpartial[i] += vpartial[i + 2];
                vpartial[i] += vpartial[i + 1];
            }


            if (i == 0)
                y[0] = - partial[0] / len;
        }
    }

    void kernel_softmax(rt::Node* node)
    {
        std::size_t rows = node->len1;
        std::size_t cols = node->len2;
        softmax1<<<rows, BLOCK_SIZE>>>(node->in1, node->out1, rows, cols);
    }

    void kernel_log_softmax(rt::Node* node)
    {
        std::size_t rows = node->len1;
        std::size_t cols = node->len2;
        log_softmax1<<<rows, BLOCK_SIZE>>>(node->in1, node->out1, rows, cols);
    }

    void kernel_softmax_cross_entropy(rt::Node* node)
    {
        std::size_t rows = node->len1;
        std::size_t cols = node->len2;
        const dbl_t* y = node->in1;
        const dbl_t* x = node->in2;
        dbl_t* out = node->out1;

        dbl_t* tmp;
        hipMalloc(&tmp, rows * sizeof(dbl_t));
        softmax_lcost1<<<rows, BLOCK_SIZE>>>(y, x, tmp, rows, cols);
        hipDeviceSynchronize();
        softmax_reduce<<<1, BLOCK_SIZE>>>(tmp, out, rows);
        hipDeviceSynchronize();
        hipFree(tmp);
    }

}
