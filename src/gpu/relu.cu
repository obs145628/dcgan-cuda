#include "hip/hip_runtime.h"
#include "kernels.hh"
#include <math_functions.h>
#include "../runtime/node.hh"



namespace gpu
{

    namespace
    {

        __device__
        dbl_t relu(dbl_t x)
        {
            return max(dbl_t(0), x);
        }


        __device__
        dbl_t relu_leaky(dbl_t x, dbl_t alpha)
        {
            return x < 0 ? alpha * x : x;
        }

        __global__
        void vect_relu(const dbl_t* x, dbl_t* y, std::size_t len)
        {
            std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
            std::size_t stride = blockDim.x * gridDim.x;

            for (std::size_t i = index; i < len; i += stride)
                y[i] = relu(x[i]);
        }

        __global__
        void vect_relu_leaky(const dbl_t* x, dbl_t* y, dbl_t alpha, std::size_t len)
        {
            std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
            std::size_t stride = blockDim.x * gridDim.x;

            for (std::size_t i = index; i < len; i += stride)
                y[i] = relu_leaky(x[i], alpha);
        }
    }
        

    void kernel_relu(rt::Node* node)
    {
        std::size_t len = node->len1;
        std::size_t block_size = 256;
        std::size_t nb_blocks = (len + block_size - 1)/ block_size;

        vect_relu<<<nb_blocks, block_size>>>(node->in1, node->out1, len);
    }

    void kernel_relu_leaky(rt::Node* node)
    {
        std::size_t len = node->len1;
        std::size_t block_size = 256;
        std::size_t nb_blocks = (len + block_size - 1)/ block_size;

        vect_relu_leaky<<<nb_blocks, block_size>>>(node->in1, node->out1,
                                                   node->alpha_leaky, len);
    }

}
