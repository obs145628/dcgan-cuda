#include "alloc.hh"
#include "mode.hh"

#include <stdexcept>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
       throw std::runtime_error {"GPUAssert: " + std::string(hipGetErrorString(code)) + " "
               + std::string(file) + ":" + std::to_string(line)};
   }
}

dbl_t* tensor_alloc(std::size_t size)
{
    if (program_mode() == ProgramMode::GPU)
    {   
        dbl_t* res;
        gpuErrchk(hipMalloc(&res, size * sizeof(dbl_t)));   
        return res;
    }
    else
        return new dbl_t[size];
}

void tensor_free(dbl_t* ptr)
{
    if (program_mode() == ProgramMode::GPU)
    {
        gpuErrchk(hipFree(ptr));
    }
    else
        delete[] ptr;

    
}
